#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cfloat>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "mnist.hpp"

#define CHECK_CUDA(expression) \
{ \
    hipError_t error = (expression); \
    if(error != 0) \
    { \
        std::cerr << "Error on line " << __LINE__ << ": " \
            << hipGetErrorString(error) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
}

void device_to_host_and_print(int height, int width, float* d_A)
{
    size_t mat_size = sizeof(float) * height * width;
    float* h_A = (float*)malloc(mat_size);
    hipMemcpy(h_A, d_A, mat_size, hipMemcpyDeviceToHost);
    for(int i = 0; i < height; ++i)
    {
        printf("[");
        for(int k = 0; k < width; ++k)
        {
            printf("%f ", h_A[i*width + k]);
        }
        printf("]\n");
    }
    free(h_A);
}

void device_to_host_and_print_int(int height, int width, int64_t* d_A)
{
    size_t mat_size = sizeof(int64_t) * height * width;
    int64_t* h_A = (int64_t*)malloc(mat_size);
    hipMemcpy(h_A, d_A, mat_size, hipMemcpyDeviceToHost);
    for(int i = 0; i < height; ++i)
    {
        printf("[");
        for(int k = 0; k < width; ++k)
        {
            printf("%ld ", h_A[i*width + k]);
        }
        printf("]\n");
    }
    free(h_A);
}

struct  mlp_t
{
    float* fc1_w;
    float* fc1_b;
    float* fc2_w;
    float* fc2_b;
    
    float* input;
    float* fc1_w_inter;
    float* fc1_b_inter;
    float* relu_inter;
    float* fc2_w_inter;
    float* fc2_b_inter;
    float* probs;
    float* ce_losses;
    float* avg_loss;

    float* dL_dce;
    float* dL_dprobs;
    float* dL_dlogits;
    float* dL_dfc2_bias;

    int64_t* labels;
};

// TODO: I think there are incorrect memory indices in this kernel and need to double check them.
template<int tile_width>
__global__ void fc_forward_kernel(
    const float* W, // Shape: (input_dim, output_dim)
    const float* X, // Shape: (batch_size, input_dim)
    float* Y,       // Shape: (batch_size, output_dim)
    int input_dim, int output_dim, int batch_size
){
    __shared__ float X_s[tile_width][tile_width];
    __shared__ float W_s[tile_width][tile_width];

    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;

    int row = block_y * tile_width + thread_y;
    int col = block_x * tile_width + thread_x;

    float Y_val = 0.0f;
    for(int ph = 0; ph < ceil(output_dim/(float)tile_width); ++ph)
    {
        // Load W tile into shared memory.
        if(row < output_dim && ph*tile_width + thread_x < input_dim)
        {
            // Tiled vertically.
            W_s[thread_y][thread_x] = W[(ph*tile_width + thread_y)*output_dim + col];
        }
        else
        {
            W_s[thread_y][thread_x] = 0.0f;
        }

        // Load X tile into shared memory.
        if(col < input_dim && ph*tile_width + thread_y < output_dim)
        {
            // Tiled horizontally. 
            X_s[thread_y][thread_x] = X[row*input_dim + ph*tile_width + thread_x];
        }
        else
        {
            X_s[thread_y][thread_x] = 0.0f;
        }
        __syncthreads();
    
        // Inner loop dot product.
        for(int k = 0; k < tile_width; ++k)
        {
            Y_val += X_s[thread_y][k] * W_s[k][thread_x];
        }
        __syncthreads();
    }

    if(row < batch_size && col < output_dim)
    {
        Y[row*output_dim + col] = Y_val;
    }
}

template<int tile_width> 
void fc_forward_launch(
    const float* W, const float* X, float* Y,
    int input_dim, int output_dim, int batch_size 
){
    const int block_size = 32;
    dim3 grid_dim((output_dim + tile_width - 1) / tile_width, (batch_size + tile_width - 1) / tile_width);
    dim3 block_dim(tile_width, tile_width);
    fc_forward_kernel<tile_width><<<grid_dim, block_dim>>>(W, X, Y, input_dim, output_dim, batch_size);
}

__global__ void bias_forward_kernel(const float* B, const float* X, float* Y, int input_dim, int batch_size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(col < input_dim && row < batch_size)
    {
        int idx = row*input_dim + col;
        Y[row*input_dim + col] = X[idx] + B[col];
    }
}

void bias_forward_launch(const float* B, const float* X, float* Y, int input_dim, int batch_size)
{
    const int block_x = (input_dim + 31) / 32;
    dim3 grid_dim((input_dim + block_x - 1) / block_x, batch_size);
    dim3 block_dim(block_x, 1);
    bias_forward_kernel<<<grid_dim, block_dim>>>(B, X, Y, input_dim, batch_size);
}

__global__ void relu_forward_kernel(const float* X, float* Y, int input_dim, int batch_size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(col < input_dim && row < batch_size)
    {
        int idx = row*input_dim + col;
        float X_val = X[idx];
        Y[row*input_dim + col] = (X_val > 0.0f) ? X_val : 0.0f;
    }
}

void relu_forward_launch(const float* X, float* Y, int input_dim, int batch_size)
{
    const int block_x = (input_dim + 31) / 32;
    dim3 grid_dim((input_dim + block_x - 1) / block_x, batch_size);
    dim3 block_dim(block_x, 1);
    relu_forward_kernel<<<grid_dim, block_dim>>>(X, Y, input_dim, batch_size);
}

template<int rows_per_block, int input_dim, int batch_size>
__global__ void softmax_forward_kernel(const float* X, float* Y)
{
    constexpr int elements_per_block = rows_per_block * input_dim;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_idx = idx / input_dim;

    __shared__ float X_s[elements_per_block];
    __shared__ float row_max[rows_per_block];
    __shared__ float row_exp_sum[rows_per_block];
    
    if(idx < input_dim * batch_size && threadIdx.x < elements_per_block)
    {
        X_s[threadIdx.x] = X[idx];
    }
    __syncthreads();

    // Turn off all threads except for the ones mapped to the start of each row.
    if(idx % input_dim == 0 && row_idx < batch_size)
    {
        // Find each row's maxmimum value. 
        row_max[row_idx] = -FLT_MAX;
        for(int i = 0; i < input_dim; ++i)
        {
            float X_val = X_s[row_idx * input_dim + i];
            if(row_max[row_idx] < X_val)
            {
                row_max[row_idx] = X_val;
            }
        }
        
        // Calculate the row's sum of exponentials.
        row_exp_sum[row_idx] = 0.0f;
        for(int i = 0; i < input_dim; ++i)
        {
            row_exp_sum[row_idx] += __expf(X_s[row_idx * input_dim + i] - row_max[row_idx]);
        }
    }
    __syncthreads();

    if(idx < input_dim * batch_size && threadIdx.x < elements_per_block)
    {
        Y[idx] = __expf(X_s[threadIdx.x] - row_max[row_idx]) / row_exp_sum[row_idx];
    }
}

template<int input_dim, int batch_size>
void softmax_forward_launch(const float* X, float* Y)
{
    constexpr int rows_per_block = (6 > batch_size) ? batch_size: 6;
    dim3 grid_dim((batch_size + rows_per_block - 1) / rows_per_block);
    dim3 block_dim(ceil((rows_per_block * input_dim) / (float)32) * 32);
    softmax_forward_kernel<rows_per_block, input_dim, batch_size><<<grid_dim, block_dim>>>(X, Y);
}

__global__ void cross_entropy_forward_kernel(
    const float* X, const int64_t* T, float* Y, int n_classes, int batch_size
){
    constexpr float eps = 0.00001f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < batch_size)
    {
        Y[idx] = -1.0f * __logf(X[idx * n_classes + T[idx]] + eps);
    }
}

void cross_entropy_forward_launch(const float* X, const int64_t* T, float* Y, int n_classes, int batch_size)
{
    dim3 grid_dim(1);
    dim3 block_dim(ceil(batch_size / (float)32) * 32);
    cross_entropy_forward_kernel<<<grid_dim, block_dim>>>(X, T, Y, n_classes, batch_size);
}

__global__ void average_forward_kernel(const float* X, float* Y, int n_inputs)
{
    if(threadIdx.x == 0 && blockIdx.x ==0)
    {
        // Sequential sum for simplicity because we're expecting a small number of inputs.
        float sum = 0.0f;
        for(int i = 0; i < n_inputs; ++i)
        {
            sum += X[i];
        }
        *Y = sum / (float)n_inputs;
    }
}

void average_forward_launch(const float* X, float* Y, int n_inputs)
{
    // We have to launch a whole warp, but we're only using one thread.
    average_forward_kernel<<<1, 32>>>(X, Y, n_inputs);
}

__global__ void average_backward_kernel(float* dL_dX, int n_inputs)
{
    if(threadIdx.x == 0 && blockIdx.x == 0)
    {
        float partial_deriv = 1.0f / (float)n_inputs;
        for(int i = 0; i < n_inputs; ++i)
        {
            dL_dX[i] = partial_deriv;
        }
    }
}

void average_backward_launch(float* dL_dX, int n_inputs)
{
    // We have to launch a whole warp, but we're only using one thread.
    average_backward_kernel<<<1, 32>>>(dL_dX, n_inputs);
}

// NOTE: not using dL_dce right now since it seems like a pretty simple kernel fusion.
// Maybe remove the average_backward_kernel.
__global__ void cross_entropy_backward_kernel(
    const float* dL_dce, const float* probs, const int64_t* labels, 
    float* dL_dprobs, int n_classes, int batch_size
){
    constexpr float eps = 0.000001f;
    constexpr float max_grad = 30.0f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n_classes * batch_size)
    {
        int label = labels[idx / n_classes];
        float val = 0.0f;
        if(idx % n_classes == label)
        {
            float prob = fmaxf(probs[idx], eps);
            val = -1.0f / ((float)batch_size * prob);
            val = fmaxf(fminf(val, max_grad), -max_grad);
        }
        dL_dprobs[idx] = val;
    }
}

void cross_entropy_backward_launch(
    const float* dL_dce, const float* probs, const int64_t* labels, 
    float* dL_dprobs, int n_classes, int batch_size
){
    const int block_x = ceil((n_classes * batch_size) / (float)32) * 32;
    //printf("block x %d\n", block_x);
    cross_entropy_backward_kernel<<<1, block_x>>>(dL_dce, probs, labels, dL_dprobs, n_classes, batch_size);
}

__global__ void softmax_backward_kernel(
    const float* probs, const int64_t* labels, float* dL_dlogits, int n_classes, int batch_size
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n_classes * batch_size)
    {
        float label_term = (labels[idx / n_classes] == idx % n_classes) ? 1.0f : 0.0f;
        dL_dlogits[idx] = probs[idx] - label_term;
    }
}

void softmax_backward_launch(
    const float* probs, const int64_t* labels, float* dL_dlogits, int n_classes, int batch_size
){
    const int block_x = ceil((n_classes * batch_size) / (float)32) * 32;
    printf("softamx back block_x %d\n", block_x);
    softmax_backward_kernel<<<1, block_x>>>(probs, labels, dL_dlogits, n_classes, batch_size);
}

__global__ void bias_out_backward_kernel(
    const float* dL_dlogits, float* dL_dbias, int n_classes, int batch_size
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n_classes * batch_size)
    {
        dL_dbias[idx] = dL_dlogits[idx];
    }
}

void bias_out_backward_launch(const float* dL_dlogits, float* dL_dbias, int n_classes, int batch_size)
{
    const int block_x = ceil((n_classes * batch_size) / (float)32) * 32;
    bias_out_backward_kernel<<<1, block_x>>>(dL_dlogits, dL_dbias, n_classes, batch_size);
}

template<int tile_width, int input_dim, int hidden_dim, int output_dim, int batch_size> 
void forward_pass(mlp_t* mlp)
{
    printf("labels:\n");
    device_to_host_and_print_int(batch_size, 1, mlp->labels);
    fc_forward_launch<tile_width>(
        (const float*)mlp->fc1_w, (const float*)mlp->input, mlp->fc1_w_inter,
        input_dim, hidden_dim, batch_size
    );
    printf("labels:\n");
    device_to_host_and_print_int(batch_size, 1, mlp->labels);

    bias_forward_launch(
        (const float*)mlp->fc1_b, (const float*)mlp->fc1_w_inter, mlp->fc1_b_inter, 
        hidden_dim, batch_size
    );
    relu_forward_launch(
        (const float*)mlp->fc1_w_inter, mlp->relu_inter, hidden_dim, batch_size
    );

    /*printf("fc1_w_inter:\n");
    device_to_host_and_print(batch_size, hidden_dim, mlp->fc1_w_inter);
    printf("\n");
    printf("fc1_b_inter:\n");
    device_to_host_and_print(batch_size, hidden_dim, mlp->fc1_b_inter);
    printf("\n");
    printf("relu_inter:\n");
    device_to_host_and_print(batch_size, hidden_dim, mlp->relu_inter);
    printf("\n");*/

    fc_forward_launch<tile_width>(
        (const float*)mlp->fc2_w, (const float*)mlp->relu_inter, mlp->fc2_w_inter,
        hidden_dim, output_dim, batch_size
    );
    bias_forward_launch(
        (const float*)mlp->fc2_b, (const float*)mlp->fc2_w_inter, mlp->fc2_b_inter, 
        output_dim, batch_size
    );
    softmax_forward_launch<output_dim, batch_size>((const float*)mlp->fc2_b_inter, mlp->probs);
    cross_entropy_forward_launch(
        (const float*)mlp->probs, (const int64_t*)mlp->labels, mlp->ce_losses, 10, batch_size
    );
    average_forward_launch((const float*)mlp->ce_losses, mlp->avg_loss, batch_size);
    
    printf("fc2_b_inter:\n");
    device_to_host_and_print(batch_size, output_dim, mlp->fc2_b_inter);
    printf("probs:\n");
    device_to_host_and_print(batch_size, output_dim, mlp->probs);
    printf("ce_losses:\n");
    device_to_host_and_print(batch_size, 1, mlp->ce_losses);
    printf("avg_loss:\n");
    device_to_host_and_print(1, 1, mlp->avg_loss); 
}

template<int tile_width, int input_dim, int hidden_dim, int output_dim, int batch_size>
void backward_pass(mlp_t* mlp)
{
    average_backward_launch(mlp->dL_dce, batch_size);
    cross_entropy_backward_launch(
        (const float*)mlp->dL_dce, (const float*)mlp->probs, (const int64_t*)mlp->labels, 
        mlp->dL_dprobs, output_dim, batch_size
    );
    softmax_backward_launch(
        (const float*)mlp->probs, (const int64_t*)mlp->labels, mlp->dL_dlogits, output_dim, batch_size
    );
    bias_out_backward_launch(
        (const float*)mlp->dL_dlogits, mlp->dL_dfc2_bias, output_dim, batch_size
    );

    printf("dL_dce:\n");
    device_to_host_and_print(batch_size, 1, mlp->dL_dce);
    printf("dL_dprobs:\n");
    device_to_host_and_print(batch_size, output_dim, mlp->dL_dprobs);
    printf("dL_dlogits:\n");
    device_to_host_and_print(batch_size, output_dim, mlp->dL_dlogits);
    printf("dL_dfc2_bias:\n");
    device_to_host_and_print(batch_size, output_dim, mlp->dL_dfc2_bias);
}

// Initialize weights to random values following a normal distribution.
__global__ void random_normal_init_kernel(float* A, int n_elements, unsigned long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < n_elements)
    {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        A[idx] = hiprand_normal(&state); 
    } 
}

void random_normal_init(int height, int width, float* A, unsigned long seed)
{
    const int n_elements = height * width;
    const int block_dim = 1024;
    const int grid_dim = (n_elements + block_dim - 1) / block_dim;
    random_normal_init_kernel<<<grid_dim, block_dim>>>(A, n_elements, seed);
}

__global__ void zero_init_kernel(float* A, int n_elements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < n_elements)
    {
        A[idx] = 0.0f;
    }
}

void zero_init(int height, int width, float* A)
{
    const int n_elements = height * width;
    const int block_dim = 1024;
    const int grid_dim = (n_elements + block_dim - 1) / block_dim;
    zero_init_kernel<<<grid_dim, block_dim>>>(A, n_elements);
}

int main()
{
    load_mnist();
    printf("Loaded MNIST\n");
    print_image(train_image[2]);

    constexpr int input_dim = 784;
    constexpr int hidden_dim = 256;
    constexpr int output_dim = 10;
    constexpr int batch_size = 4;
    constexpr int tile_width = 32;

    mlp_t mlp;
    CHECK_CUDA(hipMalloc(&mlp.fc1_w, sizeof(float) * input_dim * hidden_dim));
    CHECK_CUDA(hipMalloc(&mlp.fc2_w, sizeof(float) * hidden_dim * output_dim));
    CHECK_CUDA(hipMalloc(&mlp.fc1_b, sizeof(float) * hidden_dim));
    CHECK_CUDA(hipMalloc(&mlp.fc2_b, sizeof(float) * output_dim));
    CHECK_CUDA(hipMalloc(&mlp.input, sizeof(float) * batch_size * input_dim));
    CHECK_CUDA(hipMalloc(&mlp.fc1_w_inter, sizeof(float) * batch_size * hidden_dim));
    CHECK_CUDA(hipMalloc(&mlp.fc1_b_inter, sizeof(float) * batch_size * hidden_dim));
    CHECK_CUDA(hipMalloc(&mlp.relu_inter, sizeof(float) * batch_size * hidden_dim));
    CHECK_CUDA(hipMalloc(&mlp.fc2_w_inter, sizeof(float) * batch_size * output_dim));
    CHECK_CUDA(hipMalloc(&mlp.fc2_b_inter, sizeof(float) * batch_size * output_dim));
    CHECK_CUDA(hipMalloc(&mlp.probs, sizeof(float) * batch_size * output_dim));
    CHECK_CUDA(hipMalloc(&mlp.ce_losses, sizeof(float) * batch_size));
    CHECK_CUDA(hipMalloc(&mlp.labels, sizeof(int64_t) * batch_size));
    CHECK_CUDA(hipMalloc(&mlp.avg_loss, sizeof(float)));
    CHECK_CUDA(hipMalloc(&mlp.dL_dce, sizeof(float) * batch_size));
    CHECK_CUDA(hipMalloc(&mlp.dL_dprobs, sizeof(float) * batch_size * output_dim));
    CHECK_CUDA(hipMalloc(&mlp.dL_dlogits, sizeof(float) * batch_size * output_dim));
    CHECK_CUDA(hipMalloc(&mlp.dL_dfc2_bias, sizeof(float) * batch_size * output_dim));

    // Initialize weights and biases.
    random_normal_init(hidden_dim, input_dim, mlp.fc1_w, 0);
    random_normal_init(hidden_dim, output_dim, mlp.fc2_w, 0);
    zero_init(1, hidden_dim, mlp.fc1_b);
    zero_init(1, output_dim, mlp.fc2_b);
    zero_init(1, output_dim, mlp.probs);
    printf("Initialized weights and biases\n");


    int batch_start_idx = 0;
    // Cast batch labels to int64_t.
    int64_t batch_labels[batch_size];
    for(int i = 0; i < batch_size; ++i)
    {
        batch_labels[i] = (int64_t)train_label[batch_start_idx + i];
        printf("%d\n", batch_labels[i]);
    }
    hipMemcpy(
        mlp.input, &train_image[batch_start_idx], 
        sizeof(float) * input_dim * batch_size, hipMemcpyHostToDevice
    );
    hipMemcpy(
        mlp.labels, batch_labels, 
        sizeof(int64_t) * batch_size, hipMemcpyHostToDevice
    );
    
    forward_pass<tile_width, input_dim, hidden_dim, output_dim, batch_size>(&mlp);
    backward_pass<tile_width, input_dim, hidden_dim, output_dim, batch_size>(&mlp);
    hipFree(mlp.fc1_w);
    hipFree(mlp.fc1_b);
    hipFree(mlp.fc2_w);
    hipFree(mlp.fc2_b);
    hipFree(mlp.input);
    hipFree(mlp.fc1_w_inter);
    hipFree(mlp.fc1_b_inter);
    hipFree(mlp.relu_inter);
    hipFree(mlp.fc2_w_inter);
    hipFree(mlp.fc2_b_inter);
    hipFree(mlp.probs);
    hipFree(mlp.ce_losses);
    hipFree(mlp.avg_loss);
    hipFree(mlp.dL_dce);
    hipFree(mlp.dL_dprobs);
    hipFree(mlp.dL_dlogits);
}
